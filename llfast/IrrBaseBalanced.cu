#include "hip/hip_runtime.h"
/**
* IrrBaseBalanced.cu -- globals and kernel code to do operations on balanced radix numbers
*   This uses the Crandall irrational base method
*
* A. Thall & A. Hegedus
* Project:  gpuLucas
* 11/6/2010
*
****************************************************************************
*
* Copyright (c) 2010-2012, Andrew Thall, Alma College
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*     * Redistributions of source code must retain the above copyright
*       notice, this list of conditions and the following disclaimer.
*     * Redistributions in binary form must reproduce the above copyright
*       notice, this list of conditions and the following disclaimer in the
*       documentation and/or other materials provided with the distribution.
*     * Neither the names of Andrew Thall or Alma College, nor the
*       names of its contributors may be used to endorse or promote products
*       derived from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
* ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
* WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL ANDREW THALL OR ALMA COLLEGE BE LIABLE FOR ANY
* DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
* (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
* LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
* ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*
******************************************************************************
*
* MODIFICATIONS:
*  8/1/2011:
*     xxAH:  llintToIrrBal() now template function
*     xxAT:  Removed all dependencies on CUDPP and fast carry addition
*  2/19/2012:  xxAT release version
*/

#ifndef D_IRRBASEBALANCED
#define D_IRRBASEBALANCED

/**
 * Distribute product-accumulated bits to subsequent digits of variable base product
 * @template-param number - number of subsequent digits to distribute product bits
 */
template <int number>
static __global__ void llintToIrrBal(int *iArr, int *hiArr, long long int *lliArr, unsigned char *bperW8arr, const int size) {
	const int tid = blockIdx.x*blockDim.x + threadIdx.x;
	const int tba = threadIdx.x; // thread block address for digits index

	// Use int for each element, the radix place and its n preceeding
	__shared__ long long int digits[T_PER_B + number];
	__shared__ int signs[T_PER_B + number];
	
	// first n threads of block initialize leading digits.
	//   Be carefule to wrap-around from end of array if tid < n
	//   otherwise, load end of previous block at [tid - n]
	if (tba < number) {
     	int offset;
		if (tid - number < 0) 
			offset = size + tid;
		else
			offset = tid;
		digits[tba] = lliArr[offset - number];
		signs[tba] = digits[tba] < 0 ? -1 : 1;
		digits[tba] *= signs[tba];
	}

	digits[tba + number] = lliArr[tid];
	signs[tba + number] = digits[tba + number] < 0 ? -1 : 1;
	digits[tba + number] *= signs[tba + number]; 
	
	__syncthreads();
	
	unsigned char bperW8 = bperW8arr[tid];

	// get info for this digit
	int isHi = bperW8 & 1;
	int BITS = LO_BITS + isHi;
	int myBase = BASE_LO << isHi;
	int myMask = myBase - 1;

	// Walk backwards through the cached long longs, pulling off
	//   higher and higher order bits, all of length myMask for the
	//   current digit.
	// sbitN is amount to shift word (tid - N) before pulling off
	//   higher order bits with myMask for current digit
	int sbits1, sbits2, sbits3, sbits4, sbits5, sbits6;
	if (number >= 1) 
		sbits1 =          LO_BITS + ((bperW8 >> 1) & 1);
	if (number >= 2) 
		sbits2 = sbits1 + LO_BITS + ((bperW8 >> 2) & 1);
	if (number >= 3) 
		sbits3 = sbits2 + LO_BITS + ((bperW8 >> 3) & 1);
	if (number >= 4) 
		sbits4 = sbits3 + LO_BITS + ((bperW8 >> 4) & 1);
	if (number >= 5) 
		sbits5 = sbits4 + LO_BITS + ((bperW8 >> 5) & 1);
	if (number >= 6) 
		sbits6 = sbits5 + LO_BITS + ((bperW8 >> 6) & 1);

	int sum = signs[tba + number]*(digits[tba + number]              & myMask);
	if(number >= 1)
		sum += signs[tba + number - 1]*((digits[tba + number - 1] >> sbits1) & myMask);
	if(number >= 2)
		sum += signs[tba + number - 2]*((digits[tba + number - 2] >> sbits2) & myMask);
	if(number >= 3)
		sum += signs[tba + number - 3]*((digits[tba + number - 3] >> sbits3) & myMask);
	if(number >= 4)
		sum += signs[tba + number - 4]*((digits[tba + number - 4] >> sbits4) & myMask);
	if(number >= 5)
		sum += signs[tba + number - 5]*((digits[tba + number - 5] >> sbits5) & myMask);
	if(number >= 6)
		sum += signs[tba + number - 6]*((digits[tba + number - 6] >> sbits6) & myMask);

  /* OLD VERSION.  above really doesn't buy much.  below is simpler,
        but not templated.
  	int shiftBits = 0;
	for (int i = 1; i < 6 + 1; i++) {
		bperW8 >>= 1;
		isHi = bperW8 & 1;
		shiftBits += LO_BITS + isHi;
		sum += signs[tba + 6 - i]*((digits[tba + 6 - i] >> shiftBits) &  myMask);
	}
    */

	// do pseudo-rebalance, storing borrow or carry to hiArr[tid]
	int baseOver2 = myBase >> 1;
	int hival = 0;
	if (sum < -baseOver2)
		hival = -((-sum + baseOver2) >> BITS); //  /myBase);
	else if (sum >= baseOver2)
		hival = (sum + baseOver2) >> BITS; // /myBase;

	iArr[tid] = sum - (hival << BITS);
	hiArr[tid] = hival;
}

/**
 * do a single carry of the high-order carry of the previous digit to the
 *    current digit.  Don't rebalance if exceeds max or min on balanced
 *    representation.
 */
static __global__ void addPseudoBalanced(int *signal, int *hiAdd, int size) {

	const int tid = blockIdx.x*blockDim.x + threadIdx.x;

	if (tid == 0) 
		signal[tid] += hiAdd[size - 1];
	else
		signal[tid] += hiAdd[tid - 1];
}

/**
 * Final rebalance of irrational base representation, by one-time-only sequential
 *   add-with-carry with rebalancedIrrIntSEQGPU<<<1, 1>>> call.  Could as easily be
 *   done CPU-side.
 */
static __global__ void rebalanceIrrIntSEQGPU(int *signal, unsigned char *bpwArr, int size) {

	int carryOut = 0;
	int tBase, tBaseOver2;
	int BASE_HIOVER2 = BASE_HI >> 1;
	int BASE_LOOVER2 = BASE_LO >> 1;

	for (int i = 0; i < size; i++) {

		if (bpwArr[i] & 1) {
			tBase = BASE_HI;
			tBaseOver2 = BASE_HIOVER2;
		}
		else {
			tBase = BASE_LO;
			tBaseOver2 = BASE_LOOVER2;
		}
		int b = signal[i];

		int total = b + carryOut;

		if (total >= tBaseOver2) {
			signal[i] = total - tBase;
			carryOut = 1;
		}
		else if (total < -tBaseOver2) {
			signal[i] = total + tBase;
			carryOut = -1;
		}
		else {
			signal[i] = total;
			carryOut = 0;
		}
	}
	if (carryOut != 0)
		signal[0] += carryOut;
}

#endif // #ifndef D_IRRBASEBALANCED
